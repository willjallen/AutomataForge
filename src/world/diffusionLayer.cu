#include "hip/hip_runtime.h"
#include <vector>
#include <hip/hip_runtime.h>
#include "diffusionLayer.h"


/**
 * Updates the diffusion model on an m x n grid.
 *
 * The discretized diffusion equation is given by:
 * u_{i,j}^{n+1} = u_{i,j}^n + D * (u_{i-1,j}^n + u_{i+1,j}^n + u_{i,j-1}^n + u_{i,j+1}^n - 4 * u_{i,j}^n) + s_{i,j}
 *
 * where:
 * u_{i,j}^n represents the value at grid point (i, j) at time n,
 * D is the diffusion coefficient,
 * s_{i,j} is a source term at (i, j).
 *
 * @param u         Pointer to the current state of the grid
 * @param u_new     Pointer to the updated state of the grid
 * @param sources   Pointer to the sources grid
 * @param m         Number of rows in the grid
 * @param n         Number of columns in the grid
 * @param D         Diffusion coefficient
 */
__global__ void update_diffusion(float* u, float* u_new, const float* sources, int m, int n, float D) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i > 0 && i < m-1 && j > 0 && j < n-1) {
        float diff = D * (u[(i-1)*n + j] + u[(i+1)*n + j] + u[i*n + (j-1)] + u[i*n + (j+1)] - 4 * u[i*n + j]);
        u_new[i*n + j] = u[i*n + j] + diff + sources[i*n + j];
    }
}
